#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "pseudoGsol_cuda.h"



// function called inside collectPseudoGsol

int pseudoGsol::call_to_kern(int offsetStatic, double *transI, double *pGsol, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE *staticQPointsOctree, QPOINT *staticQPoints, int numStaticQPoints, int numStaticQPointsOctreeNodes, PG *movingPG, double TRANSLATEstatic, double DIMstatic, int rangeCountStatic, PARAMS_IN params, int offsetMoving, double *trans, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE *movingQPointsOctree, QPOINT *movingQPoints, int numMovingQPoints, int numMovingQPointsOctreeNodes, PG *staticPG, double TRANSLATEmoving, double DIMmoving, int rangeCountMoving)
{

// convert movingGrid to array

		vector<ball_cuda> ballsVecMoving_h = convertGrid(movingPG);

		int totBallsMoving = ballsVecMoving_h.size();
		int *totBallsMoving_d;

		int totSizeMoving = totBallsMoving * (sizeof(int)*3+sizeof(Point));

		ball_cuda *arrayMoving_h;
		hipHostAlloc( (void**)&arrayMoving_h, totSizeMoving, hipHostMallocDefault);\

		std::copy(ballsVecMoving_h.begin(), ballsVecMoving_h.end(), arrayMoving_h);
		ball_cuda *arrayMoving_d;	


// convert staticGrid to array

		vector<ball_cuda> ballsVecStatic_h = convertGrid(staticPG);

		int totBallsStatic = ballsVecStatic_h.size();
		int *totBallsStatic_d;

		int totSizeStatic = totBallsStatic * (sizeof(int)*3+sizeof(Point));

		ball_cuda *arrayStatic_h;
		hipHostAlloc( (void**)&arrayStatic_h, totSizeStatic, hipHostMallocDefault);

		std::copy(ballsVecStatic_h.begin(), ballsVecStatic_h.end(), arrayStatic_h);
		ball_cuda *arrayStatic_d;



// build grids --------------------------------------------

		hipStream_t streamS, streamM;
		hipStreamCreate(&streamS);
		hipStreamCreate(&streamM);

		hipDeviceSetLimit (hipLimitMallocHeapSize, 512*1024*1024);

		hipMalloc( (void **)&arrayMoving_d, totSizeMoving );
		hipMalloc( (void **)&totBallsMoving_d, sizeof(int) );

		hipMalloc( (void **)&arrayStatic_d, totSizeStatic );
		hipMalloc( (void **)&totBallsStatic_d, sizeof(int) );

		hipMemcpyAsync( arrayMoving_d, arrayMoving_h, totSizeMoving, hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( totBallsMoving_d, &totBallsMoving, sizeof(int), hipMemcpyHostToDevice, streamS );

		hipMemcpyAsync( arrayStatic_d, arrayStatic_h, totSizeStatic, hipMemcpyHostToDevice, streamM );
		hipMemcpyAsync( totBallsStatic_d, &totBallsStatic, sizeof(int), hipMemcpyHostToDevice, streamM );
	
		

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error1: %s\n", hipGetErrorString(err));

		
		copyMovingGridtoGPU<<< 1, 1, 0, streamS >>>(arrayMoving_d, totBallsMoving_d);

 		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error2: %s\n", hipGetErrorString(err));


		copyStaticGridtoGPU<<< 1, 1, 0, streamM >>>(arrayStatic_d, totBallsStatic_d);



		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error3: %s\n", hipGetErrorString(err));


		hipFree(arrayMoving_d);
		hipFree(totBallsMoving_d);
		hipFree(arrayStatic_d);
		hipFree(totBallsStatic_d);

		hipHostFree(arrayMoving_h);
		hipHostFree(arrayStatic_h);


// end building grids---------------------------------------------------



// passing structs

// QPOINT

		struct QPOINT_CUDA *staticQPoints_h, *staticQPoints_d;
		hipHostAlloc( (void**)&staticQPoints_h, numStaticQPoints * sizeof(QPOINT_CUDA), hipHostMallocDefault);

		for (int i = 0; i < numStaticQPoints; i++)
		{
			staticQPoints_h[i].x = staticQPoints[i].x;
			staticQPoints_h[i].y = staticQPoints[i].y;
			staticQPoints_h[i].z = staticQPoints[i].z;
			staticQPoints_h[i].w = staticQPoints[i].w;
			staticQPoints_h[i].h = staticQPoints[i].h;
		}			
		

		struct QPOINT_CUDA *movingQPoints_h, *movingQPoints_d;
		hipHostAlloc( (void**)&movingQPoints_h, numMovingQPoints * sizeof(QPOINT_CUDA), hipHostMallocDefault);

		for (int i = 0; i < numMovingQPoints; i++)
		{
			movingQPoints_h[i].x = movingQPoints[i].x;
			movingQPoints_h[i].y = movingQPoints[i].y;
			movingQPoints_h[i].z = movingQPoints[i].z;
			movingQPoints_h[i].w = movingQPoints[i].w;
			movingQPoints_h[i].h = movingQPoints[i].h;
		}
		

// QPOINTS_OCTREE_NODE

		struct QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree_h, *staticQPointsOctree_d;
		hipHostAlloc( (void**)&staticQPointsOctree_h, numStaticQPointsOctreeNodes * sizeof(QPOINTS_OCTREE_NODE_CUDA), hipHostMallocDefault);


		for (int i = 0; i < numStaticQPointsOctreeNodes; i++)
		{
			staticQPointsOctree_h[i].qPtsStartID = staticQPointsOctree[i].qPtsStartID;
			staticQPointsOctree_h[i].qPtsEndID = staticQPointsOctree[i].qPtsEndID;
		}

		struct QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree_h, *movingQPointsOctree_d;
		hipHostAlloc( (void**)&movingQPointsOctree_h, numMovingQPointsOctreeNodes * sizeof(QPOINTS_OCTREE_NODE_CUDA), hipHostMallocDefault);


		for (int i = 0; i < numMovingQPointsOctreeNodes; i++)
		{
			movingQPointsOctree_h[i].qPtsStartID = movingQPointsOctree[i].qPtsStartID;
			movingQPointsOctree_h[i].qPtsEndID = movingQPointsOctree[i].qPtsEndID;
		}

// others

		struct static_params *statS_h, *statS_d;
		hipHostAlloc( (void**)&statS_h, 3*(sizeof(int)+sizeof(double)), hipHostMallocDefault);

		statS_h->offset = offsetStatic;
		statS_h->numStaticQPointsOctreeNodes = numStaticQPointsOctreeNodes;
		statS_h->distanceCutoff = params.distanceCutoff;
		statS_h->TRANSLATE = TRANSLATEstatic;
		statS_h->DIM = DIMstatic;
		statS_h->rangeCount = rangeCountStatic;


		struct moving_params *statM_h, *statM_d;
		hipHostAlloc( (void**)&statM_h, 3*(sizeof(int)+sizeof(double)), hipHostMallocDefault);

		statM_h->offset = offsetMoving;
		statM_h->numMovingQPointsOctreeNodes = numMovingQPointsOctreeNodes;
		statM_h->distanceCutoff = params.distanceCutoff;
		statM_h->TRANSLATE = TRANSLATEmoving;
		statM_h->DIM = DIMmoving;
		statM_h->rangeCount = rangeCountMoving;


		bool *staticQPointsOctreeFlags_d, *movingQPointsOctreeFlags_d;
		double *trans_d, *transI_d;

		double *pGsolStatic_d, *pGsolMoving_d, *pGsolStatic_h, *pGsolMoving_h;
		double *pGsolHStaticPos_d, *pGsolHStaticNeg_d, *pGsolHStaticPos_h, *pGsolHStaticNeg_h;
		double *pGsolHMovingPos_d, *pGsolHMovingNeg_d, *pGsolHMovingPos_h, *pGsolHMovingNeg_h;




// allocate arrays to return the results

		hipHostAlloc( (void**)&pGsolStatic_h, sizeof(double) * 4*128, hipHostMallocDefault);
		hipHostAlloc( (void**)&pGsolMoving_h, sizeof(double) * 4*128, hipHostMallocDefault);

		hipHostAlloc( (void**)&pGsolHStaticPos_h, sizeof(double) * 4*128, hipHostMallocDefault);
		hipHostAlloc( (void**)&pGsolHStaticNeg_h, sizeof(double) * 4*128, hipHostMallocDefault);

		hipHostAlloc( (void**)&pGsolHMovingPos_h, sizeof(double) * 4*128, hipHostMallocDefault);
		hipHostAlloc( (void**)&pGsolHMovingNeg_h, sizeof(double) * 4*128, hipHostMallocDefault);



// allocate vairables to return the results, in case the reduction is done inside the GPU

		*pGsol = *pGsolHStaticPos = *pGsolHStaticNeg = *pGsolHMovingPos = *pGsolHMovingNeg = 0;

		/*double *pGsolStatic_scalar_d, *pGsolMoving_scalar_d, *pGsolHStaticPos_scalar_d, *pGsolHStaticNeg_scalar_d, *pGsolHMovingPos_scalar_d, *pGsolHMovingNeg_scalar_d;
		
		double *pGsolStatic_scalar_h, *pGsolMoving_scalar_h;

		hipHostAlloc( (void**)&pGsolStatic_scalar_h, sizeof(double), hipHostMallocDefault);
		hipHostAlloc( (void**)&pGsolMoving_scalar_h, sizeof(double), hipHostMallocDefault);
*/

		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error1: %s\n", hipGetErrorString(err));



// hipMalloc ---------------------------


		hipMalloc( (void **)&staticQPoints_d, sizeof(QPOINT_CUDA)*numStaticQPoints );
		hipMalloc( (void **)&movingQPoints_d, sizeof(QPOINT_CUDA)*numMovingQPoints );

		hipMalloc( (void **)&staticQPointsOctree_d, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numStaticQPointsOctreeNodes );
		hipMalloc( (void **)&movingQPointsOctree_d, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numMovingQPointsOctreeNodes );

		hipMalloc( (void **)&staticQPointsOctreeFlags_d, sizeof(bool)*numStaticQPointsOctreeNodes );
		hipMalloc( (void **)&movingQPointsOctreeFlags_d, sizeof(bool)*numMovingQPointsOctreeNodes );

		hipMalloc( (void **)&trans_d, sizeof(double)*12 );
		hipMalloc( (void **)&transI_d, sizeof(double)*12 );

		hipMalloc( (void **)&statS_d, 3*(sizeof(int)+sizeof(double)) );//sizeof(static_params) );
		hipMalloc( (void **)&statM_d, 3*(sizeof(int)+sizeof(double)) );


		hipMalloc( (void **)&pGsolStatic_d, sizeof(double) * 4*128);
		hipMalloc( (void **)&pGsolMoving_d, sizeof(double) * 4*128);

		hipMalloc( (void **)&pGsolHStaticPos_d, sizeof(double) * 4*128);
		hipMalloc( (void **)&pGsolHStaticNeg_d, sizeof(double) * 4*128);

		hipMalloc( (void **)&pGsolHMovingPos_d, sizeof(double) * 4*128);
		hipMalloc( (void **)&pGsolHMovingNeg_d, sizeof(double) * 4*128);



/*
		hipMalloc( (void **)&pGsolStatic_scalar_d, sizeof(double));
		hipMalloc( (void **)&pGsolMoving_scalar_d, sizeof(double));

		hipMalloc( (void **)&pGsolHStaticPos_scalar_d, sizeof(double));
		hipMalloc( (void **)&pGsolHStaticNeg_scalar_d, sizeof(double));

		hipMalloc( (void **)&pGsolHMovingPos_scalar_d, sizeof(double));
		hipMalloc( (void **)&pGsolHMovingNeg_scalar_d, sizeof(double));
*/

	
// hipMemcpy-------------------

/*
		hipMemcpyAsync( arrayMoving_d, arrayMoving_h, totSizeMoving, hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( totBallsMoving_d, &totBallsMoving, sizeof(int), hipMemcpyHostToDevice, streamS );


		hipMemcpyAsync( arrayStatic_d, arrayStatic_h, totSizeStatic, hipMemcpyHostToDevice, streamM );
		hipMemcpyAsync( totBallsStatic_d, &totBallsStatic, sizeof(int), hipMemcpyHostToDevice, streamM );
*/

		hipMemcpyAsync( staticQPoints_d, staticQPoints_h, sizeof(QPOINT_CUDA)*numStaticQPoints, hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( movingQPoints_d, movingQPoints_h, sizeof(QPOINT_CUDA)*numMovingQPoints, hipMemcpyHostToDevice, streamM );


		hipMemcpyAsync( staticQPointsOctree_d, staticQPointsOctree_h, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numStaticQPointsOctreeNodes, hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( movingQPointsOctree_d, movingQPointsOctree_h, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numMovingQPointsOctreeNodes, hipMemcpyHostToDevice, streamM );



		hipMemcpyAsync( staticQPointsOctreeFlags_d, staticQPointsOctreeFlags, sizeof(bool)*numStaticQPointsOctreeNodes, hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( movingQPointsOctreeFlags_d, movingQPointsOctreeFlags, sizeof(bool)*numMovingQPointsOctreeNodes, hipMemcpyHostToDevice, streamM );



		hipMemcpyAsync( trans_d, trans, sizeof(double)*12, hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( transI_d, transI, sizeof(double)*12, hipMemcpyHostToDevice, streamM );


		hipMemcpyAsync( statS_d, statS_h, 3*(sizeof(int)+sizeof(double)), hipMemcpyHostToDevice, streamS );
		hipMemcpyAsync( statM_d, statM_h, 3*(sizeof(int)+sizeof(double)), hipMemcpyHostToDevice, streamM );




// call Kernel ---------------------


		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error108: %s\n", hipGetErrorString(err));
);

		kern1<<<4,128, 0, streamS >>>(&(statS_d->offset), transI_d, pGsolStatic_d, pGsolHStaticPos_d, pGsolHStaticNeg_d, staticQPointsOctreeFlags_d, staticQPointsOctree_d, staticQPoints_d, &(statS_d->numStaticQPointsOctreeNodes), &(statS_d->TRANSLATE), &(statS_d->DIM), &(statS_d->rangeCount), &(statS_d->distanceCutoff));//, pGsolStatic_scalar_d, pGsolHStaticPos_scalar_d, pGsolHStaticNeg_scalar_d);

		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error2: %s\n", hipGetErrorString(err));

	kern2<<< 4,128, 0, streamM >>>(&(statS_d->distanceCutoff), &(statM_d->offset), trans_d, pGsolMoving_d, pGsolHMovingPos_d, pGsolHMovingNeg_d, movingQPointsOctreeFlags_d, movingQPointsOctree_d, movingQPoints_d, &(statM_d->numMovingQPointsOctreeNodes), &(statM_d->TRANSLATE), &(statM_d->DIM), &(statM_d->rangeCount));//, pGsolMoving_scalar_d, pGsolHMovingPos_scalar_d, pGsolHMovingNeg_scalar_d);



		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error3: %s\n", hipGetErrorString(err));


// copy back the results, either in arrays or in simple variables

/*
	hipMemcpyAsync( pGsolStatic_scalar_h, pGsolStatic_scalar_d, sizeof(double), hipMemcpyDeviceToHost, streamS );	
	hipMemcpyAsync( pGsolHStaticPos, pGsolHStaticPos_scalar_d, sizeof(double), hipMemcpyDeviceToHost, streamS );	
	hipMemcpyAsync( pGsolHStaticNeg, pGsolHStaticNeg_scalar_d, sizeof(double), hipMemcpyDeviceToHost, streamS );	

	hipMemcpyAsync( pGsolMoving_scalar_h, pGsolMoving_scalar_d, sizeof(double), hipMemcpyDeviceToHost, streamM );	
	hipMemcpyAsync( pGsolHMovingPos, pGsolHMovingPos_scalar_d, sizeof(double), hipMemcpyDeviceToHost, streamM );	
	hipMemcpyAsync( pGsolHMovingNeg, pGsolHMovingNeg_scalar_d, sizeof(double), hipMemcpyDeviceToHost, streamM );	
*/


	hipMemcpyAsync( pGsolStatic_h, pGsolStatic_d, sizeof(double) * 4*128, hipMemcpyDeviceToHost, streamS );	
	hipMemcpyAsync( pGsolHStaticPos_h, pGsolHStaticPos_d, sizeof(double) * 4*128, hipMemcpyDeviceToHost, streamS );	
	hipMemcpyAsync( pGsolHStaticNeg_h, pGsolHStaticNeg_d, sizeof(double) * 4*128, hipMemcpyDeviceToHost, streamS );	

	hipMemcpyAsync( pGsolMoving_h, pGsolMoving_d, sizeof(double) * 4*128, hipMemcpyDeviceToHost, streamM );	
	hipMemcpyAsync( pGsolHMovingPos_h, pGsolHMovingPos_d, sizeof(double) * 4*128, hipMemcpyDeviceToHost, streamM );	
	hipMemcpyAsync( pGsolHMovingNeg_h, pGsolHMovingNeg_d, sizeof(double) * 4*128, hipMemcpyDeviceToHost, streamM );	


// synchronize

		hipStreamSynchronize(streamS);
		hipStreamSynchronize(streamM);

		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error3: %s\n", hipGetErrorString(err));

		for (int i = 0; i < (4*128); i++)
		{
			//printf("pGsolStatic_h[i] %lf\n", pGsolStatic_h[i]);
			*pGsol += pGsolStatic_h[i];
			*pGsolHStaticPos += pGsolHStaticPos_h[i];
			*pGsolHStaticNeg += pGsolHStaticNeg_h[i];
		}
	
		for (int i = 0; i < (4*128); i++)
		{
			*pGsol += pGsolMoving_h[i];
			*pGsolHMovingPos += pGsolHMovingPos_h[i];
			*pGsolHMovingNeg += pGsolHMovingNeg_h[i];
		}	


	//	printf("CUDA *pGsol, *pGsolHStaticPos, *pGsolHStaticNeg, *pGsolHMovingPos, *pGsolHMovingNeg %lf %lf %lf %lf %lf\n", *pGsol, *pGsolHStaticPos, *pGsolHStaticNeg, *pGsolHMovingPos, *pGsolHMovingNeg);

	
// free memory
		
		hipStreamDestroy(streamS);
		hipStreamDestroy(streamM);

		hipFree(staticQPoints_d);
		hipFree(movingQPoints_d);

		hipFree(staticQPointsOctree_d);
		hipFree(movingQPointsOctree_d);

		hipFree(staticQPointsOctreeFlags_d);
		hipFree(movingQPointsOctreeFlags_d);

		hipFree(trans_d);
		hipFree(transI_d);

		hipFree(statS_d);
		hipFree(statM_d);

		hipFree(pGsolStatic_d);
		hipFree(pGsolMoving_d);

		hipFree(pGsolHStaticPos_d);
		hipFree(pGsolHStaticNeg_d);

		hipFree(pGsolHMovingPos_d);
		hipFree(pGsolHMovingNeg_d);

		hipFree(pGsolStatic_d);
		hipFree(pGsolMoving_d);

		hipFree(pGsolHStaticPos_d);
		hipFree(pGsolHStaticNeg_d);

		hipFree(pGsolHMovingPos_d);
		hipFree(pGsolHMovingNeg_d);
		
		hipHostFree(staticQPointsOctree_h);
		hipHostFree(staticQPoints_h);

		hipHostFree(movingQPointsOctree_h);
		hipHostFree(movingQPoints_h);

		hipHostFree(statS_h);
		hipHostFree(statM_h);

		hipHostFree(pGsolStatic_h);
		hipHostFree(pGsolMoving_h);

		hipHostFree(pGsolHStaticPos_h);
		hipHostFree(pGsolHStaticNeg_h);

		hipHostFree(pGsolHMovingPos_h);
		hipHostFree(pGsolHMovingNeg_h);

		hipDeviceReset();

	return 0;
}



// prints the grids copied to GPU
__device__ void printGrid(grid_cuda g) 
{

		plane_cuda plane;
		line_cuda line;
		gridcell_cuda cell;
		Point_cuda point;

	//	printf("CUDA PGrid nPlanes %d\n", g.nElems);


		for (int i = 0; i < g.nElems; i++) // goes through all the planes
		{	
			plane = plane_cuda(g.ptr[i].id, g.ptr[i].nElems, g.ptr[i].ptr);
			printf("CUDA plane ID %d\n", plane.id);
		//	printf("CUDA nLines %d\n", plane.nElems);
				
			for (int j = 0; j < plane.nElems; j++) // goes through all the lines
			{
				line =  line_cuda(plane.ptr[j].id, plane.ptr[j].nElems, plane.ptr[j].ptr);
				printf("	CUDA line ID %d\n", line.id);
		//		printf("CUDA nCells %d\n", line.nElems);
				
				for (int k = 0; k < line.nElems; k++) // goes through all the gridcells
				{
					cell = gridcell_cuda(line.ptr[k].id, line.ptr[k].nElems, line.ptr[k].ptr);
					printf("		CUDA cell ID %d\n", cell.id);
					printf("			CUDA nBalls %d\n", cell.nElems);
					
					for (int l = 0; l < cell.nElems; l++) // goes through all the Points
					{
						point = Point_cuda(cell.ptr[l].x, cell.ptr[l].y, cell.ptr[l].z);
//						printf("CUDA Point xx %f\n", point.x);
//						printf("CUDA Point yy %f\n", point.y);
//						printf("CUDA Point zz %f\n", point.z);
					} // end points
					cell.freePtr();

				} // end cells
				line.freePtr();

			} // end lines
			plane.freePtr();

		}	// end planes

	
}


// constructs the staticGrid on GPU, is invoked from the host (CPU)

__global__ void copyStaticGridtoGPU(ball_cuda *balls, int *totBalls) //OK
{

		int curCellID, curLineID, curPlaneID;

		// to store temporary lists of planes, lines, an cells
		dynArray_cuda<plane_cuda> planes = dynArray_cuda<plane_cuda>(10); 	
		dynArray_cuda<line_cuda> lines;
		dynArray_cuda<gridcell_cuda> cells;
		dynArray_cuda<Point> particles;

		gridcell_cuda cell;
		line_cuda line;
		plane_cuda plane;


		int i = 0;
		curCellID =  balls[i].cell_id;
		curLineID =  balls[i].parentLine;
		curPlaneID =  balls[i].parentPlane;

 	
		while ( i < *totBalls )
		{
//			printf("CUDA curPlaneID %d\n", curPlaneID);
//			printf("CUDA i %d\n", i);

			lines = dynArray_cuda<line_cuda>(5);
			while( balls[i].parentPlane == curPlaneID )		
			{

				cells = dynArray_cuda<gridcell_cuda>(5);
				while ( balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
				{

					particles = dynArray_cuda<Point>(10);		
					while ( balls[i].cell_id == curCellID && balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
					{
						particles.push_end(balls[i].ball);
						i++;	
					}	// end cells		

	//				printf("CUDA cell id %d\n", curCellID);
//					printf("CUDA nBalls %d\n", particles.getSize());
					
					cell = gridcell_cuda(curCellID, particles.getSize(), particles.getAll());
					cells.push_end(cell);
					particles.freeElements();
					curCellID =  balls[i].cell_id;
				} // end lines

//				printf("CUDA line id %d\n", curLineID);
//				printf("CUDA nCells %d\n", cells.getSize());

				line = line_cuda(curLineID, cells.getSize(), cells.getAll());
				lines.push_end(line);
				cells.freeElements();
				curLineID =  balls[i].parentLine;
			} // end planes

//			printf("CUDA plane id %d\n", curPlaneID);
//			printf("CUDA nLines %d\n", lines.getSize());

			plane = plane_cuda(curPlaneID, lines.getSize(), lines.getAll());
		//	printf("curPlaneID plane id %d %d\n", curPlaneID, plane.id);
			planes.push_end(plane);		
			lines.freeElements();
			curPlaneID =  balls[i].parentPlane;
		} // end all

		staticGrid = grid_cuda(planes.getSize(), planes.getAll());
		planes.freeElements();

		//printGrid(staticGrid);

}



// constructs the movingGrid on GPU, is invoked from the host (CPU)

__global__ void copyMovingGridtoGPU(ball_cuda *balls, int *totBalls) 
{
		int curCellID, curLineID, curPlaneID;

		// to store temporary lists of planes, lines, an cells
		dynArray_cuda<plane_cuda> planes = dynArray_cuda<plane_cuda>(10); 	
		dynArray_cuda<line_cuda> lines;
		dynArray_cuda<gridcell_cuda> cells;
		dynArray_cuda<Point> particles;

		gridcell_cuda cell;
		line_cuda line;
		plane_cuda plane;


		int i = 0;
		curCellID =  balls[i].cell_id;
		curLineID =  balls[i].parentLine;
		curPlaneID =  balls[i].parentPlane;

 	
		while ( i < *totBalls )
		{
//			printf("CUDA curPlaneID %d\n", curPlaneID);
//			printf("CUDA i %d\n", i);

			lines = dynArray_cuda<line_cuda>(5);
			while( balls[i].parentPlane == curPlaneID )		
			{

				cells = dynArray_cuda<gridcell_cuda>(5);
				while ( balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
				{

					particles = dynArray_cuda<Point>(10);		
					while ( balls[i].cell_id == curCellID && balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
					{
						particles.push_end(balls[i].ball);
						i++;	
					}	// end cells		

	//				printf("CUDA cell id %d\n", curCellID);
//					printf("CUDA nBalls %d\n", particles.getSize());
					
					cell = gridcell_cuda(curCellID, particles.getSize(), particles.getAll());
					cells.push_end(cell);
					particles.freeElements();
					curCellID =  balls[i].cell_id;
				} // end lines

//				printf("CUDA line id %d\n", curLineID);
//				printf("CUDA nCells %d\n", cells.getSize());

				line = line_cuda(curLineID, cells.getSize(), cells.getAll());
				lines.push_end(line);
				cells.freeElements();
				curLineID =  balls[i].parentLine;
			} // end planes

//			printf("CUDA plane id %d\n", curPlaneID);
//			printf("CUDA nLines %d\n", lines.getSize());

			plane = plane_cuda(curPlaneID, lines.getSize(), lines.getAll());
		//	printf("curPlaneID plane id %d %d\n", curPlaneID, plane.id);
			planes.push_end(plane);		
			lines.freeElements();
			curPlaneID =  balls[i].parentPlane;
		} // end all

		movingGrid = grid_cuda(planes.getSize(), planes.getAll());
		planes.freeElements();

		//printGrid(movingGrid);

}



// is invoked fromt he device (GPU) and constructs a grid

__device__ grid_cuda copyGrid(ball_cuda *balls, int *totBalls) 
{
		int curCellID, curLineID, curPlaneID;

		// to store temporary lists of planes, lines, an cells
		dynArray_cuda<plane_cuda> planes = dynArray_cuda<plane_cuda>(10); 	
		dynArray_cuda<line_cuda> lines;
		dynArray_cuda<gridcell_cuda> cells;
		dynArray_cuda<Point> particles;

		gridcell_cuda cell;
		line_cuda line;
		plane_cuda plane;


		int i = 0;
		curCellID =  balls[i].cell_id;
		curLineID =  balls[i].parentLine;
		curPlaneID =  balls[i].parentPlane;

 	
		while ( i < *totBalls )
		{
//			printf("CUDA curPlaneID %d\n", curPlaneID);
//			printf("CUDA i %d\n", i);

			lines = dynArray_cuda<line_cuda>(5);
			while( balls[i].parentPlane == curPlaneID )		
			{

				cells = dynArray_cuda<gridcell_cuda>(5);
				while ( balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
				{

					particles = dynArray_cuda<Point>(10);		
					while ( balls[i].cell_id == curCellID && balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
					{
						particles.push_end(balls[i].ball);
						i++;	
					}	// end cells		

	//				printf("CUDA cell id %d\n", curCellID);
//					printf("CUDA nBalls %d\n", particles.getSize());
					
					cell = gridcell_cuda(curCellID, particles.getSize(), particles.getAll());
					cells.push_end(cell);
					particles.freeElements();
					curCellID =  balls[i].cell_id;
				} // end lines

//				printf("CUDA line id %d\n", curLineID);
//				printf("CUDA nCells %d\n", cells.getSize());

				line = line_cuda(curLineID, cells.getSize(), cells.getAll());
				lines.push_end(line);
				cells.freeElements();
				curLineID =  balls[i].parentLine;
			} // end planes

//			printf("CUDA plane id %d\n", curPlaneID);
//			printf("CUDA nLines %d\n", lines.getSize());

			plane = plane_cuda(curPlaneID, lines.getSize(), lines.getAll());
		//	printf("curPlaneID plane id %d %d\n", curPlaneID, plane.id);
			planes.push_end(plane);		
			lines.freeElements();
			curPlaneID =  balls[i].parentPlane;
		} // end all
	
		grid_cuda g = grid_cuda(planes.getSize(), planes.getAll());


		planes.freeElements();
		return g;

}




// implements the CPU function distsq on the GPU

__device__ float distsq_cuda(Point_cuda *a, Point_cuda *b) 
{ 
		float res;
		double dx, dy, dz;

		dx = a->x - b->x;
		dy = a->y - b->y;
		dz = a->z - b->z;  
		res = dx*dx + dy*dy + dz*dz;

		return res;
}


// implements the CPU function transformPoint in the GPU
__device__ void transformPoint_cuda( Point_cuda *p, double *transMat, Point_cuda *np ) 
{
   np->x = transMat[  0 ] * p->x + transMat[  1 ] * p->y + transMat[  2 ] * p->z + transMat[  3 ];
   np->y = transMat[  4 ] * p->x + transMat[  5 ] * p->y + transMat[  6 ] * p->z + transMat[  7 ];
   np->z = transMat[  8 ] * p->x + transMat[  9 ] * p->y + transMat[ 10 ] * p->z + transMat[ 11 ];       
}


//implements the CPU function pointsWithinRange in the GPU
__device__ bool pointsWithinRange_cuda(Point_cuda *q, double *delta, double *TRANSLATE, double *DIM, int *rangeCount, grid_cuda *g)
{

  Point_cuda p;
	p.x = q->x;
	p.y = q->y;
	p.z = q->z;

  p.x += *TRANSLATE;
  p.y += *TRANSLATE;
  p.z += *TRANSLATE;

  int l = (int)((p.z - (*delta)) / (*DIM));
  int h = (int)((p.z + (*delta)) / (*DIM));


  int size,size1,size2;
  
  dynArray_cuda<line_cuda>  temp1;
  dynArray_cuda<gridcell_cuda> temp2;

	dynArray_cuda<plane_cuda> S2;
	S2 = g->report_d(l,h);

  (*rangeCount)++;

  if(S2.empty())
    return false;

  size2 = S2.getSize();


  for(int i = 0; i < size2; i++) 
  {
    (*rangeCount)++;
   
    l = (int)((p.y - (*delta))/ (*DIM));
    h = (int)((p.y + (*delta))/ (*DIM));
    temp1 = (S2.get(i)).report_d(l,h);
    
    size1 = temp1.getSize();


    for(int j = 0; j < size1; j++) 
    {
      (*rangeCount)++;

      l = (int)((p.x - (*delta))/ (*DIM));
      h = (int)((p.x + (*delta))/ (*DIM));


      if(!(temp1.get(j).empty()))
      {
				temp2 = (temp1.get(j)).report_d(l,h);
						  	
				size = temp2.getSize();

				Point_cuda *oa = (Point_cuda*) malloc (sizeof (Point_cuda));

				double delsq = (*delta)*(*delta);
				for(int mn=0;mn<size;mn++)
				{
					int atomsInCell = temp2.get(mn).getSize();
					for(int pq=0;pq<atomsInCell;pq++)
					{
						oa->x = temp2.get(mn).ptr[pq].x;
						oa->y = temp2.get(mn).ptr[pq].y;
						oa->z = temp2.get(mn).ptr[pq].z;;
						float dist = (distsq_cuda(oa, q));
		
						
						if(distsq_cuda(oa, q) <= delsq)
							return true;
					}
				}
				free(oa);
      	temp2.freeElements();
      }
    }
    temp1.freeElements();
  }
	S2.freeElements();

  return false;
}




// implements the first part of collectPseudoGsol in GPU
__device__ void pseudoGsolStatic(int *offset, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes,  double *TRANSLATE, double *DIM, int *rangeCount, double *distanceCutoff)
{


	int i = blockDim.x*blockIdx.x+threadIdx.x;

	pGsolStatic[i] = 0;
	pGsolHStaticPos[i] = 0;
	pGsolHStaticNeg[i] = 0;

	if (i >= 0 && i < *numStaticQPointsOctreeNodes)
	{
		if ( staticQPointsOctreeFlags[ *offset + i ])
		{

		for( int j = staticQPointsOctree[ i ].qPtsStartID; j <= staticQPointsOctree[ i ].qPtsEndID; j++ )
			{	
				
				Point_cuda p, q;
               
        p.x = staticQPoints[ j ].x;
        p.y = staticQPoints[ j ].y;
        p.z = staticQPoints[ j ].z;

   	    transformPoint_cuda( &p, transI, &q );

    	  if ( pointsWithinRange_cuda( &q, distanceCutoff, TRANSLATE, DIM, rangeCount, &movingGrid) )
				{
        	( pGsolStatic[i] ) += staticQPoints[ j ].w;

          if ( staticQPoints[ j ].h > 0 ) 
						( pGsolHStaticPos[i] ) += staticQPoints[ j ].h * staticQPoints[ j ].w;
          else 
						( pGsolHStaticNeg[i] ) += staticQPoints[ j ].h * staticQPoints[ j ].w;
      	}  
      }            
		}
	}
	
}



// implements the second part of collectPseudoGsol in GPU
__device__ void pseudoGsolMoving(int *offset, double *trans, double *pGsolMoving, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree, QPOINT_CUDA *movingQPoints, int *numMovingQPointsOctreeNodes, double *distanceCutoff, double *TRANSLATE, double *DIM, int *rangeCount)
{

	int i = blockDim.x*blockIdx.x+threadIdx.x;


	pGsolMoving[i] = 0;
	pGsolHMovingPos[i] = 0;
	pGsolHMovingNeg[i] = 0;

	if (i >= 0 && i < *numMovingQPointsOctreeNodes)
	{
		if ( movingQPointsOctreeFlags[ *offset + i ])
		{
			for (int j = movingQPointsOctree[ i ].qPtsStartID;  j <= movingQPointsOctree[ i ].qPtsEndID; j++ )
			{
				Point_cuda p, q;
               
        p.x = movingQPoints[ j ].x;
        p.y = movingQPoints[ j ].y;
        p.z = movingQPoints[ j ].z;
               
        transformPoint_cuda( &p, trans, &q );
           
        if ( pointsWithinRange_cuda( &q, distanceCutoff, TRANSLATE, DIM, rangeCount, &staticGrid ) )
        {
        	( pGsolMoving[i] ) += movingQPoints[ j ].w;

          if ( movingQPoints[ j ].h > 0 ) 
						( pGsolHMovingPos[i] ) += movingQPoints[ j ].h * movingQPoints[ j ].w;
          else 
						( pGsolHMovingNeg[i] ) += movingQPoints[ j ].h * movingQPoints[ j ].w;
        }  
     	}            
		}
	}

}



// both of the next 2 functions are used to do a reduction in CUDA, presently they only do it for threads in the same block, to do the reduction among ALL the threads they have to be modified

__device__ void warpReduce(volatile double *sdata, unsigned int tid) 
{
		int blockSize = 512;
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

//template <unsigned int blockSize>
__device__ void reduce4(double *g_idata, double *g_odata, int n) 
{
		__shared__ double sdata[512];
		unsigned int tid = threadIdx.x;

		// each thread loads one element from global to shared mem

		if ( tid < n)
			sdata[tid] = g_idata[tid];	
		else
				sdata[tid] = 0;

		__syncthreads();

		// do reduction in shared mem
		for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
			if (tid < s ){
				sdata[tid] += sdata[tid + s];
			}
			__syncthreads();
		}


		if (tid == 0){
			*g_odata = sdata[0];
		}

}




// the only reason why both of the next 2 function were created instead of calling the pseudoGsol functions directly was so that some pre/pos-processing could be done without making the code "messy"

__global__ void kern1(/*static stuff*/int *offsetStatic, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes,  double *TRANSLATEstatic, double *DIMstatic, int *rangeCountStatic, double *distanceCutoff)// /*others*/, double *pGsolStatic_scalar_d, double *pGsolHStaticPos_scalar_d, double *pGsolHStaticNeg_scalar_d)
{
/*
		__shared__ grid_cuda movingGrid;

		if (threadIdx.x == 0)
			movingGrid = copyGrid(ballsStatic, totBallsStatic);

			__syncthreads();
*/
			pseudoGsolStatic(offsetStatic, transI, pGsolStatic, pGsolHStaticPos, pGsolHStaticNeg, staticQPointsOctreeFlags, staticQPointsOctree, staticQPoints, numStaticQPointsOctreeNodes, TRANSLATEstatic, DIMstatic, rangeCountStatic, distanceCutoff);

/*
		reduce4(pGsolStatic, pGsolStatic_scalar_d, *numStaticQPointsOctreeNodes);
		reduce4(pGsolHStaticPos, pGsolHStaticPos_scalar_d, *numStaticQPointsOctreeNodes);
		reduce4(pGsolHStaticNeg, pGsolHStaticNeg_scalar_d, *numStaticQPointsOctreeNodes);
*/
}


__global__ void kern2(double *distanceCutoff /*moving stuff*/,int *offsetMoving, double *trans, double *pGsolMoving, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree, QPOINT_CUDA *movingQPoints, int *numMovingQPointsOctreeNodes,  double *TRANSLATEmoving, double *DIMmoving, int *rangeCountMoving)// /*others*/, double *pGsolMoving_scalar_d, double *pGsolHMovingPos_scalar_d, double *pGsolHMovingNeg_scalar_d)
{

	/*
		__shared__	grid_cuda staticGrid;

		if (threadIdx.x == 0)
			staticGrid = copyGrid(ballsMoving, totBallsMoving);

			__syncthreads();
*/
			pseudoGsolMoving(offsetMoving, trans, pGsolMoving, pGsolHMovingPos, pGsolHMovingNeg, movingQPointsOctreeFlags, movingQPointsOctree, movingQPoints, numMovingQPointsOctreeNodes, distanceCutoff, TRANSLATEmoving, DIMmoving, rangeCountMoving);
/*
		reduce4(pGsolMoving, pGsolMoving_scalar_d, *numMovingQPointsOctreeNodes);
		reduce4(pGsolHMovingPos, pGsolHMovingPos_scalar_d, *numMovingQPointsOctreeNodes);
		reduce4(pGsolHMovingNeg, pGsolHMovingNeg_scalar_d, *numMovingQPointsOctreeNodes);
*/
}




// converts the CPU grid to a vector which contains all the points with parents plane, line, and gridcell IDs 'attached'

vector<ball_cuda> convertGrid(PG *grid_)
{

		int nPlanes;
		int nLines;
		int nCells;
		int nBalls;

		int grid_init, grid_end;	
		int plane_init, plane_end;
		int line_init, line_end;
		
		vector<tuple<plane*> > grid1;

		tuple<plane*> plane1;
		vector<tuple<line*> > plane2;

		tuple<line*> line1;
		vector<tuple<gridcell*> > line2;
		
		tuple<gridcell*> cell1;
		
		vector<ball_cuda> ballsVec_h;
		ball_cuda tempBall_h;

		grid g1 = grid_->getGrid();

		nPlanes = g1.RR.getn();


		grid_init = g1.RR.getOverallMin();
		grid_end = g1.RR.getOverallMax();	
		grid1 = g1.RR.report(grid_init, grid_end);

		// create vector with total number of balls and plane, line, and cell ID
		
		for (int i = 0; i < nPlanes; i++)  // goes through all planes
		{
			plane1 = grid1.at(i);
	//		printf("plane ID %d\n", plane1.id);

			nLines = (plane1.ptr)->RR.getn();
//			printf(" nLines %d\n", nLines);
			
			plane_init =(plane1.ptr)->RR.getOverallMin(); 
			plane_end = (plane1.ptr)->RR.getOverallMax();

			plane2 = (plane1.ptr)->RR.report(plane_init, plane_end);

			
			for (int j = 0; j < nLines; j++)  // goes through all lines
			{
				line1 = plane2.at(j);
	//			printf("	line ID %d\n", line1.id);

				nCells = (line1.ptr)->RR.getn();
		//		printf("nCells  %d\n", nCells);

				line_init = (line1.ptr)->RR.getOverallMin(); 
				line_end = (line1.ptr)->RR.getOverallMax(); 
				line2 = (line1.ptr)->RR.report(line_init, line_end);

				
				for (int k = 0; k < nCells; k++)  // goes through all cells
				{
					cell1 = line2.at(k);
	//				printf("		cell ID %d\n", cell1.id);
					

					nBalls = cell1.ptr->balls.size();
	//				printf("			nBalls %d\n", nBalls);
	
					for (int kk = 0 ; kk < nBalls; kk++)  //goes through all balls
					{
						tempBall_h.cell_id = cell1.id;
						tempBall_h.parentLine = line1.id;
						tempBall_h.parentPlane = plane1.id;
						tempBall_h.ball = Point (cell1.ptr->balls.at(kk)->x, cell1.ptr->balls.at(kk)->y, cell1.ptr->balls.at(kk)->z);

						ballsVec_h.push_back(tempBall_h);

					} //end balls	
				} // end cells
			} // end lines
		} // end planes
		
		return ballsVec_h;

}
