#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//#include "dynArray_cpu.h"

#include "../../inc/pseudoGsol.h"
#include "pseudoGsol_cuda.h"



__device__ void printGrid(grid_cuda g)
{

		plane_cuda plane;
		line_cuda line;
		gridcell_cuda cell;
		Point_cuda point;

		plane_cuda *planes2;
		line_cuda *lines2;
		gridcell_cuda *cells2;


		printf("CUDA PGrid nPlanes %d\n", g.nElems);
		planes2 = g.report_d(g.ptr[g.nElems-1].id, g.ptr[0].id);
		/*
		for (int i  = 0; i < g.nElems; i++)
			printf("CUDA plane ID %d\n", g.ptr[i].id);
		*/

		for (int i = 0; i < g.nElems; i++)
		{	
			plane = plane_cuda(g.ptr[i].id, g.ptr[i].nElems, g.ptr[i].ptr);
			lines2 = plane.report_d(plane.ptr[plane.nElems-1].id, plane.ptr[0].id); 
			printf("CUDA plane ID %d\n", plane.id);
			printf("CUDA nLines %d\n", plane.nElems);
				
			for (int j = 0; j < plane.nElems; j++)
			{
				line =  line_cuda(plane.ptr[j].id, plane.ptr[j].nElems, plane.ptr[j].ptr);
				cells2 = line.report_d(line.ptr[line.nElems-1].id, line.ptr[0].id);
				printf("CUDA line ID %d\n", line.id);
				printf("CUDA nCells %d\n", line.nElems);				
								
				for (int k = 0; k < line.nElems; k++)
				{
					cell = gridcell_cuda(line.ptr[k].id, line.ptr[k].nElems, line.ptr[k].ptr);
					printf("CUDA cell ID %d\n", cell.id);
					printf("CUDA nBalls %d\n", cell.nElems);
					
					for (int l = 0; l < cell.nElems; l++)
					{
						point = Point_cuda(cell.ptr[l].x, cell.ptr[l].y, cell.ptr[l].z);
//						printf("CUDA Point xx %f\n", point.x);
//						printf("CUDA Point yy %f\n", point.y);
//						printf("CUDA Point zz %f\n", point.z);
					}
//					cell.freePtr();
				}
//				line.freePtr();
			}
//			plane.freePtr();
		}	


//		plane.freePtr();
//		line.freePtr();
//		cell.freePtr();

		free(planes2);
		free(lines2);
		free(cells2);
		
}


__device__ grid_cuda copyGrid(ball_cuda *balls, int *totBalls) //OK
{

//		printf("ola2\n");
		int curCellID, curLineID, curPlaneID;

		// to store temporary lists of planes, lines, an cells
		dynArray_cuda<plane_cuda> planes = dynArray_cuda<plane_cuda>(10); 	
		dynArray_cuda<line_cuda> lines;//  = dynArray_cuda<line_cuda>(); 	
		dynArray_cuda<gridcell_cuda> cells;// = dynArray_cuda<gridcell_cuda>(); 		
		dynArray_cuda<Point> particles;// = dynArray_cuda<Point>(); 		

		gridcell_cuda cell;
		line_cuda line;
		plane_cuda plane;


		int i = 0;
		curCellID =  balls[i].cell_id;
		curLineID =  balls[i].parentLine;
		curPlaneID =  balls[i].parentPlane;

 	
		while ( i < *totBalls )
		{
//			printf("CUDA curPlaneID %d\n", curPlaneID);
//			printf("CUDA i %d\n", i);

			lines = dynArray_cuda<line_cuda>(10);
			while( balls[i].parentPlane == curPlaneID )		
			{

				cells = dynArray_cuda<gridcell_cuda>(10);
				while ( balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
				{

					particles = dynArray_cuda<Point>(10);		
					while ( balls[i].cell_id == curCellID && balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
					{
						particles.push_end(balls[i].ball);
						i++;	
					}	// end cells		

//					printf("CUDA cell id %d\n", curCellID);
//					printf("CUDA nBalls %d\n", particles.getSize());
					
					cell = gridcell_cuda(curCellID, particles.getSize(), particles.getAll());
					cells.push_end(cell);
//					particles.freeElements();
//					cell.freePtr();
					curCellID =  balls[i].cell_id;
				} // end lines

//				printf("CUDA line id %d\n", curLineID);
//				printf("CUDA nCells %d\n", cells.getSize());

				line = line_cuda(curLineID, cells.getSize(), cells.getAll());
				lines.push_end(line);
//				cells.freeElements();
//				line.freePtr();
				curLineID =  balls[i].parentLine;
			} // end planes

//			printf("CUDA plane id %d\n", curPlaneID);
//			printf("CUDA nLines %d\n", lines.getSize());

			plane = plane_cuda(curPlaneID, lines.getSize(), lines.getAll());
			planes.push_end(plane);		
//			lines.freeElements();
//			plane.freePtr();
			curPlaneID =  balls[i].parentPlane;
		} // end all
	
		grid_cuda g = grid_cuda(planes.getSize(), planes.getAll());

		printf("CUDA nPlanes %d\n", planes.getSize());

//		planes.freeElements();
			

		return g;

}

/*


__device__ float distsq_cuda(Point a, Point b) {
		float res;
		double dx, dy, dz;

		dx = a.x - b.x;
		dy = a.y - b.y;
		dz = a.z - b.z;  
		res = dx*dx + dy*dy + dz*dz;

		return res;
}



__device__ void transformPoint_cuda( Point p, double *transMat, Point *np )
{
   np->x = transMat[  0 ] * p.x + transMat[  1 ] * p.y + transMat[  2 ] * p.z + transMat[  3 ];
   np->y = transMat[  4 ] * p.x + transMat[  5 ] * p.y + transMat[  6 ] * p.z + transMat[  7 ];
   np->z = transMat[  8 ] * p.x + transMat[  9 ] * p.y + transMat[ 10 ] * p.z + transMat[ 11 ];       
}



__device__ bool pointsWithinRange_cuda(Point *q, double delta, double TRANSLATE, double DIM, int rangeCount, grid g)
{
  Point p = *q;

  p.x += TRANSLATE;
  p.y += TRANSLATE;
  p.z += TRANSLATE;

  int l = (int)(( p.z - delta) / DIM);
  int h = (int)((p.z + delta) / DIM);

  int i,j,k,size,size1,size2,m;
  
  vector<tuple<line*> >  temp1;
  vector<tuple<gridcell*> >  temp2, S0, S1;
  vector<tuple<gridcell*> >::iterator start2, end2; 
  
  int tts;


  vector<tuple<plane*> > S2 = g.RR.report(l,h);
  rangeCount++;

  if(S2.empty())
  {
    return false;
  }
  size2 = (int)S2.size();


  for(i = 0; i < size2; i++) 
  {
    rangeCount++;
   
    l = (int)((p.y - delta)/DIM);
    h = (int)((p.y + delta)/DIM);
    temp1 = (S2[i].ptr)->RR.report(l,h);
    
    size1 = (int) temp1.size();
    
    for(j = 0; j < size1; j++) 
    {
      rangeCount++;

      l = (int)((p.x - delta)/DIM);
      h = (int)((p.x + delta)/DIM);


      if((temp1[j].ptr)->RR.getn())
      {
				temp2 = (temp1[j].ptr)->RR.report(l,h);
						  
				size = (int)temp2.size();
				int pbcount;

				Point *oa;

				double delsq = delta*delta;
				for(int mn=0;mn<size;mn++)
				{
					int atomsInCell = temp2[mn].ptr->balls.size();
					for(int pq=0;pq<atomsInCell;pq++)
					{
						oa = temp2[mn].ptr->balls[pq];

						if(oa->distsq_cuda(*oa, *q) <= delsq) 
							return true;
					}
				}
	
      	temp2.clear();
	
      }
    }
    temp1.clear();
  }

  return false;
}







__device__ pseudoGsol_static(int offset, double *transI, double *pGsol, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE *staticQPointsOctree, QPOINT *staticQPoints, PG *movingPG, PARAMS_IN params)
{

	int nTot = blockDim * gridDim;

	int i = blockIdx.x;
	int j = threadIdx.x;	
	int ij = i* blockDim.x + j;

	if (ij >= 0 && ij < nTot)
	{
		if ( staticQPointsOctreeFlags[ offset + i ])
		{
			if ( ( blockIdx.x == i &&  j => staticQPointsOctree[ i ].qPtsStartID) && (blockIdx.x == i && j <= staticQPointsOctree[ i ].qPtsEndID)
			{
				
				Point p, q;
               
        p.x = staticQPoints[ j ].x;
        p.y = staticQPoints[ j ].y;
        p.z = staticQPoints[ j ].z;
               
        transformPoint_cuda( p, transI, &q );
               
        if ( movingPG->pointsWithinRange_cuda( &q, params.distanceCutoff ) )
        {
        	( *pGsol ) += staticQPoints[ j ].w;

          if ( staticQPoints[ j ].h > 0 ) 
						( *pGsolHStaticPos ) += staticQPoints[ j ].h * staticQPoints[ j ].w;
          else 
						( *pGsolHStaticNeg ) += staticQPoints[ j ].h * staticQPoints[ j ].w;
        }  
      }            
	
		}

	}

}


__device__ pseudoGsol_moving(int offset, double *trans, double *pGsol, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE *movingQPointsOctree, QPOINT *movingQPoints, PG *staticPG, PARAMS_IN params)
{

	int nTot = blockDim * gridDim;

	int i = blockIdx.x * blockDim.x;
	int j = threadIdx.x;	
	int ij = i*j;

	if (ij >= 0 && ij < nTot)
	{
		if ( movingQPointsOctreeFlags[ offset + i ])
		{
			if ( ( blockIdx.x == i &&  j => movingQPointsOctree[ i ].qPtsStartID) && (blockIdx.x == i && j <= movingQPointsOctree[ i ].qPtsEndID)
			{
			{
				Point p, q;
               
        p.x = movingQPoints[ j ].x;
        p.y = movingQPoints[ j ].y;
        p.z = movingQPoints[ j ].z;
               
        transformPoint_cuda( p, trans, &q );
               
        if ( staticPG->pointsWithinRange_cuda( &q, params.distanceCutoff ) )
        {
        	( *pGsol ) += movingQPoints[ j ].w;

          if ( movingQPoints[ j ].h > 0 ) 
						( *pGsolHMovingPos ) += movingQPoints[ j ].h * movingQPoints[ j ].w;
          else 
						( *pGsolHMovingNeg ) += movingQPoints[ j ].h * movingQPoints[ j ].w;
        }  
      }            
	
		}

	}

}



*/


__global__ void kern(ball_cuda *balls, int *totBalls)
{
		//printf("ola\n");

		grid_cuda g = copyGrid(balls, totBalls);
		

		
		printGrid(g);		

		g.freePtr();
/*
		printf("CUDA totBalls %d\n", *totBalls);



		printf("CUDA cell0 id %d\n", balls[0].cell_id);
		printf("CUDA line0 id %d\n", balls[0].parentLine);
		printf("CUDA plane0 id %d\n", balls[0].parentPlane);

		printf("CUDA ball0 x %f\n", balls[0].ball.x);
		printf("CUDA ball0 y %f\n", balls[0].ball.y);
		printf("CUDA ball0 z %f\n", balls[0].ball.z);

		printf("CUDA cell id %d\n", balls[(*totBalls)-1].cell_id);
		printf("CUDA line id %d\n", balls[(*totBalls)-1].parentLine);
		printf("CUDA plane id %d\n", balls[(*totBalls)-1].parentPlane);

		printf("CUDA ball x %f\n", balls[(*totBalls)-1].ball.x);
		printf("CUDA ball y %f\n", balls[(*totBalls)-1].ball.y);
		printf("CUDA ball z %f\n", balls[(*totBalls)-1].ball.z);

*/
/*
		//printf("ball000 %f\n", balls[9].ball.x);
		dynArray_cuda<Point> arr = dynArray_cuda<Point>();
		for (int i = 0; i < 100; i++){
			arr.push_end(balls[i].ball);
		}
		for (int i = 0; i < 100; i++){
			printf("arr %f\n", arr.get(i).x);
			printf("ball %f\n", balls[i].ball.x);
		}	
		
*/

}

vector<ball_cuda> convertGrid(PG *grid_)
{

		int nPlanes;
		int nLines;
		int nCells;
		int nBalls;

		int grid_init, grid_end;	
		int plane_init, plane_end;
		int line_init, line_end;
		
		vector<tuple<plane*> > grid1;

		tuple<plane*> plane1;
		vector<tuple<line*> > plane2;

		tuple<line*> line1;
		vector<tuple<gridcell*> > line2;
		
		tuple<gridcell*> cell1;
		
		vector<ball_cuda> ballsVec_h;
		ball_cuda tempBall_h;

		grid g1 = grid_->getGrid();


		nPlanes = g1.RR.getRepMap().size();
//		printf("nPlanes %d\n", nPlanes);
			
		
		grid_init = g1.RR.getRepMap().at(0);
		grid_end = g1.RR.getRepMap().at(nPlanes-1);	
		grid1 = g1.RR.report(grid_init, grid_end);
		

		// count the total number of balls
		
		// goes through all planes
		for (int i = 0; i < nPlanes; i++)
		{
			plane1 = grid1.at(i);
//			printf(" plane ID %d\n", plane1.id);

			
			nLines = (plane1.ptr)->RR.getRepMap().size();
//			printf(" nLines %d\n", nLines);
			
			plane_init =(plane1.ptr)->RR.getRepMap().at(0); 
			plane_end = (plane1.ptr)->RR.getRepMap().at(nLines-1);
			plane2 = (plane1.ptr)->RR.report(plane_init, plane_end);
			
			// goes through all lines
			for (int j = 0; j < nLines; j++)
			{
				line1 = plane2.at(j);
//				printf("line ID %d\n", line1.id);
		
				nCells = (line1.ptr)->RR.getRepMap().size();
//				printf("nCells %d\n", nCells);
				
				line_init = (line1.ptr)->RR.getRepMap().at(0); 
				line_end = (line1.ptr)->RR.getRepMap().at(nCells-1); 
				line2 = (line1.ptr)->RR.report(line_init, line_end);
				
				
				// goes through all cells
				for (int k = 0; k < nCells; k++)
				{
					cell1 = line2.at(k);
//					printf("cell ID %d\n", cell1.id);
					

					nBalls = cell1.ptr->balls.size();
//					printf("nBalls %d\n", nBalls);


					// create vector with total number of balls and plane, line, and cell ID
					//goes through all balls
					for (int kk = 0 ; kk < nBalls; kk++)
					{
					
						tempBall_h.cell_id = cell1.id;
						tempBall_h.parentLine = line1.id;
						tempBall_h.parentPlane = plane1.id;
						tempBall_h.ball = Point (cell1.ptr->balls.at(kk)->x, cell1.ptr->balls.at(kk)->y, cell1.ptr->balls.at(kk)->z);

						ballsVec_h.push_back(tempBall_h);

					} //end balls	
				} // end cells
			} // end lines
		} // end planes
		
		return ballsVec_h;

}




/*
int call_to_kern(int offset, double *transI, double *pGsol, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE *staticQPointsOctree, QPOINT *staticQPoints, PG *movingPG, PARAMS_IN params,
double *trans, double *pGsol, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE *movingQPointsOctree, QPOINT *movingQPoints, PG *staticPG,
double TRANSLATE, double DIM, int rangeCount)*/
int call_to_kern( PG *movingPG)
{

		vector<ball_cuda> ballsVec_h = convertGrid(movingPG);

		int totBalls = ballsVec_h.size();
		int *totBalls_d;



		int totSize = totBalls * (sizeof(int)*3+sizeof(Point));

//		printf("tot size %d\n", totSize);

		ball_cuda *arrayBalls_h = (ball_cuda*) malloc(totSize);
		std::copy(ballsVec_h.begin(), ballsVec_h.end(), arrayBalls_h);

		ball_cuda *arrayBalls_d;

/*

		printf("cell0 id %d\n", arrayBalls_h[0].cell_id);
		printf("line0 id %d\n", arrayBalls_h[0].parentLine);
		printf("plane0 id %d\n", arrayBalls_h[0].parentPlane);

		printf("ball0 x %f\n", arrayBalls_h[0].ball.x);
		printf("ball0 y %f\n", arrayBalls_h[0].ball.y);
		printf("ball0 z %f\n", arrayBalls_h[0].ball.z);

		printf("cell id %d\n", arrayBalls_h[totBalls-1].cell_id);
		printf("line id %d\n", arrayBalls_h[totBalls-1].parentLine);
		printf("plane id %d\n", arrayBalls_h[totBalls-1].parentPlane);

		printf("ball x %f\n", arrayBalls_h[totBalls-1].ball.x);
		printf("ball y %f\n", arrayBalls_h[totBalls-1].ball.y);
		printf("ball z %f\n", arrayBalls_h[totBalls-1].ball.z);
		
		printf("totballs %d\n", totBalls);
*/




		hipDeviceSetLimit (hipLimitMallocHeapSize, 134217728);
		hipMalloc( (void **)&arrayBalls_d, totSize );
		hipMalloc( (void **)&totBalls_d, sizeof(int) );

		hipMemcpy( arrayBalls_d, arrayBalls_h, totSize, hipMemcpyHostToDevice );
		hipMemcpy( totBalls_d, &totBalls, sizeof(int), hipMemcpyHostToDevice );
		printf("calling...\n");
		kern<<< 1,1 >>>(arrayBalls_d, totBalls_d);
		printf("end...\n");

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error: %s\n", hipGetErrorString(err));

		hipFree(arrayBalls_d);
		hipFree(totBalls_d);

		free(arrayBalls_h);

		hipDeviceReset();



	return 0;
}




