#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
//#include "dynArray_cpu.h"

#include "../../inc/pseudoGsol.h"
#include "pseudoGsol_cuda.h"

vector<ball_cuda> convertGrid(PG *grid_);
__device__ void printGrid(grid_cuda g);
__device__ grid_cuda copyGrid(ball_cuda *balls, int *totBalls);


__device__ void coisas(grid_cuda g) //OK
{

	for (int i = 0; i < g.nElems; i++)
			printf("	coisas i g.ptr[i].id g.ptr[i] g.ptr[i].nElems, %d %d %d\n", i, g.ptr[i].id, g.ptr[i].nElems);


	dynArray_cuda<plane_cuda> S2;
	S2 = g.report_d(20,20);

  int size2 = S2.getSize();

printf("CUDA coisas size2 %d\n", size2);


}

__device__ void printGrid(grid_cuda g) //OK
{

		plane_cuda plane;
		line_cuda line;
		gridcell_cuda cell;
		Point_cuda point;

	//	printf("CUDA PGrid nPlanes %d\n", g.nElems);


		for (int i = 0; i < g.nElems; i++)
		{	
		//	printf("i g.ptr[i].id, %d %d\n", i, g.ptr[i].id);
			plane = plane_cuda(g.ptr[i].id, g.ptr[i].nElems, g.ptr[i].ptr);
			printf("CUDA plane ID %d\n", plane.id);
		//	printf("CUDA nLines %d\n", plane.nElems);
				
			for (int j = 0; j < plane.nElems; j++)
			{
		//		printf("j plane.ptr[j].id %d %d\n", j, plane.ptr[j].id);
				line =  line_cuda(plane.ptr[j].id, plane.ptr[j].nElems, plane.ptr[j].ptr);
				printf("	CUDA line ID %d\n", line.id);
		//		printf("CUDA nCells %d\n", line.nElems);
				
				for (int k = 0; k < line.nElems; k++)
				{
				//	printf("k line.ptr[k].id %d %d\n", k, line.ptr[k].id);
					cell = gridcell_cuda(line.ptr[k].id, line.ptr[k].nElems, line.ptr[k].ptr);
					printf("		CUDA cell ID %d\n", cell.id);
					printf("			CUDA nBalls %d\n", cell.nElems);
					
					for (int l = 0; l < cell.nElems; l++)
					{
						point = Point_cuda(cell.ptr[l].x, cell.ptr[l].y, cell.ptr[l].z);
//						printf("CUDA Point xx %f\n", point.x);
//						printf("CUDA Point yy %f\n", point.y);
//						printf("CUDA Point zz %f\n", point.z);
					} // end points
					cell.freePtr();

				} // end cells
				line.freePtr();

			} // end lines
			plane.freePtr();

		}	// end planes

	
}


__device__ grid_cuda copyGrid(ball_cuda *balls, int *totBalls) //OK
{
		int curCellID, curLineID, curPlaneID;

		// to store temporary lists of planes, lines, an cells
		dynArray_cuda<plane_cuda> planes = dynArray_cuda<plane_cuda>(10); 	
		dynArray_cuda<line_cuda> lines;
		dynArray_cuda<gridcell_cuda> cells;
		dynArray_cuda<Point> particles;

		gridcell_cuda cell;
		line_cuda line;
		plane_cuda plane;


		int i = 0;
		curCellID =  balls[i].cell_id;
		curLineID =  balls[i].parentLine;
		curPlaneID =  balls[i].parentPlane;

 	
		while ( i < *totBalls )
		{
//			printf("CUDA curPlaneID %d\n", curPlaneID);
//			printf("CUDA i %d\n", i);

			lines = dynArray_cuda<line_cuda>(5);
			while( balls[i].parentPlane == curPlaneID )		
			{

				cells = dynArray_cuda<gridcell_cuda>(5);
				while ( balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
				{

					particles = dynArray_cuda<Point>(10);		
					while ( balls[i].cell_id == curCellID && balls[i].parentLine == curLineID && balls[i].parentPlane == curPlaneID )
					{
						particles.push_end(balls[i].ball);
						i++;	
					}	// end cells		

	//				printf("CUDA cell id %d\n", curCellID);
//					printf("CUDA nBalls %d\n", particles.getSize());
					
					cell = gridcell_cuda(curCellID, particles.getSize(), particles.getAll());
					cells.push_end(cell);
					particles.freeElements();
					curCellID =  balls[i].cell_id;
				} // end lines

//				printf("CUDA line id %d\n", curLineID);
//				printf("CUDA nCells %d\n", cells.getSize());

				line = line_cuda(curLineID, cells.getSize(), cells.getAll());
				lines.push_end(line);
				cells.freeElements();
				curLineID =  balls[i].parentLine;
			} // end planes

//			printf("CUDA plane id %d\n", curPlaneID);
//			printf("CUDA nLines %d\n", lines.getSize());

			plane = plane_cuda(curPlaneID, lines.getSize(), lines.getAll());
		//	printf("curPlaneID plane id %d %d\n", curPlaneID, plane.id);
			planes.push_end(plane);		
			lines.freeElements();
			curPlaneID =  balls[i].parentPlane;
		} // end all
	
		grid_cuda g = grid_cuda(planes.getSize(), planes.getAll());
/*
		for (int i = 0; i < planes.getSize(); i++)
			printf(" i, planes.get(i), planes.get(i).id %d %d %d\n", i, planes.get(i), planes.get(i).id);				
		

		//printf("CUDA nPlanes %d\n", planes.getSize());


		for (int i = 0; i < g.nElems; i++)
			printf(" antes i plane.id %d %d\n", i, g.ptr[i].id);
		
			
		for (int i = 0; i < g.nElems; i++)
			printf("depois i plane.id %d %d\n", i,g.ptr[i].id);
*/

		planes.freeElements();
		return g;

}


	





__device__ float distsq_cuda(Point_cuda *a, Point_cuda *b) { //should be OK
		float res;
		double dx, dy, dz;
//		printf("OLAAA\n");
		dx = a->x - b->x;
		dy = a->y - b->y;
		dz = a->z - b->z;  
		res = dx*dx + dy*dy + dz*dz;
//		printf("res %lf\n", res);
		return res;
}



__device__ void transformPoint_cuda( Point_cuda *p, double *transMat, Point_cuda *np ) //OK
{
   np->x = transMat[  0 ] * p->x + transMat[  1 ] * p->y + transMat[  2 ] * p->z + transMat[  3 ];
   np->y = transMat[  4 ] * p->x + transMat[  5 ] * p->y + transMat[  6 ] * p->z + transMat[  7 ];
   np->z = transMat[  8 ] * p->x + transMat[  9 ] * p->y + transMat[ 10 ] * p->z + transMat[ 11 ];       
}



__device__ bool pointsWithinRange_cuda(Point_cuda *q, double *delta, double *TRANSLATE, double *DIM, int *rangeCount, grid_cuda *g)
{

	//hist[threadIdx.x] += 1;
//		printf("INSIDE distanceCutoff TRANSLATE DIM rangeCount %lf %lf %lf %d\n", *delta, *TRANSLATE, *DIM, *rangeCount);
  Point_cuda p;
	p.x = q->x;
	p.y = q->y;
	p.z = q->z;

/*printf("threadIdx.x %d\n",threadIdx.x);
if (threadIdx.x == 217){
        printf("A p.x %lf\n", p.x);  
        printf("A p.y %lf\n", p.y);
        printf("A p.z %lf\n", p.z);
}*/
  p.x += *TRANSLATE;
  p.y += *TRANSLATE;
  p.z += *TRANSLATE;
/*if (threadIdx.x == 217){
        printf("D p.x %lf\n", p.x);  
        printf("D p.y %lf\n", p.y);
        printf("D p.z %lf\n", p.z);
}*/
  int l = (int)((p.z - (*delta)) / (*DIM));
  int h = (int)((p.z + (*delta)) / (*DIM));



  int size,size1,size2;
  
  dynArray_cuda<line_cuda>  temp1;
  dynArray_cuda<gridcell_cuda> temp2;


  //int tts;
//	printf("CUDA l, h %d %d\n", l, h);
	dynArray_cuda<plane_cuda> S2;
	S2 = g->report_d(l,h);

  (*rangeCount)++;

  if(S2.empty())
  {
		//printf("CUDA s2 empty \n");
    return false;
  }
  size2 = S2.getSize();

//printf("CUDA  l h size2 %d %d %d \n",l, h, size2);


//	printf("CUDA size2 %d\n", size2);

  for(int i = 0; i < size2; i++) 
  {
    (*rangeCount)++;
   
    l = (int)((p.y - (*delta))/ (*DIM));
    h = (int)((p.y + (*delta))/ (*DIM));
    temp1 = (S2.get(i)).report_d(l,h);
    
    size1 = temp1.getSize();



//    printf("CUDA size1 %d i %d\n", size1, i);
//		hist[threadIdx.x] += size1;
    for(int j = 0; j < size1; j++) 
    {
      (*rangeCount)++;

      l = (int)((p.x - (*delta))/ (*DIM));
      h = (int)((p.x + (*delta))/ (*DIM));


      if(!(temp1.get(j).empty()))
      {
				temp2 = (temp1.get(j)).report_d(l,h);
						  	
				size = temp2.getSize();
//				int pbcount;
			//	printf("CUDA  l h size %d %d %d \n",l, h, size);

				Point_cuda *oa = (Point_cuda*) malloc (sizeof (Point_cuda));
		 	  //printf("CUDA S2.get(i).id, temp1.get(j).id l h size i %d %d %d %d %d\n", S2.get(i).id, temp1.get(j).id, l, h, size, i);

				double delsq = (*delta)*(*delta);
				for(int mn=0;mn<size;mn++)
				{
					int atomsInCell = temp2.get(mn).getSize();
	//				printf("atomsInCell %d\n", atomsInCell);
					for(int pq=0;pq<atomsInCell;pq++)
					{
		//				printf("OLA111111111111111111111\n");
						//printf("temp2.get(mn).ptr[pq].x %lf\n", temp2.get(mn).ptr[pq].x);
						oa->x = temp2.get(mn).ptr[pq].x;
						oa->y = temp2.get(mn).ptr[pq].y;
						oa->z = temp2.get(mn).ptr[pq].z;
		//				printf("OLA222222222222222222222\n");
//						printf("delsq %lf\n", delsq);
						float dist = (distsq_cuda(oa, q));
		
						
						if(distsq_cuda(oa, q) <= delsq){ 
				//			printf("CUDA thread dist delsq %d %f %lf\n", threadIdx.x, dist, delsq);
							//printf("TRUE\n");
							return true;
			//				printf("OOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOOO\n");
						}
					}
				}
				free(oa);
	
      	temp2.freeElements();
	
      }
    }
    temp1.freeElements();
  }
	S2.freeElements();

  return false;
}







//__device__ void pseudoGsol_static(int *offset, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes, grid_cuda *movingPG, double *distanceCutoff, double *TRANSLATE, double *DIM, int *rangeCount)

__device__ void pseudoGsolStatic(grid_cuda *movingPG, int *offset, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes,  double *TRANSLATE, double *DIM, int *rangeCount, double *distanceCutoff)//, double *pGsol_scalar_d, double *pGsolHStaticPos_scalar_d, double *pGsolHStaticNeg_scalar_d, double *pGsolHMovingPos_scalar_d, double *pGsolHMovingNeg_scalar_d)
{


	int i = threadIdx.x;

	if (i >= 0 && i < *numStaticQPointsOctreeNodes)
	{
		pGsolStatic[i] = 0;
		pGsolHStaticPos[i] = 0;
		pGsolHStaticNeg[i] = 0;
		
		if ( staticQPointsOctreeFlags[ *offset + i ])
		{
			for (int j = staticQPointsOctree[ i ].qPtsStartID; j <= staticQPointsOctree[ i ].qPtsEndID; j++ )
			{	

				Point_cuda p, q;
               
        p.x = staticQPoints[ j ].x;
        p.y = staticQPoints[ j ].y;
        p.z = staticQPoints[ j ].z;

   	    transformPoint_cuda( &p, transI, &q );

    	   if ( pointsWithinRange_cuda( &q, distanceCutoff, TRANSLATE, DIM, rangeCount, movingPG) )
				{
        	( pGsolStatic[i] ) += staticQPoints[ j ].w;

          if ( staticQPoints[ j ].h > 0 ) 
						( pGsolHStaticPos[i] ) += staticQPoints[ j ].h * staticQPoints[ j ].w;
          else 
						( pGsolHStaticNeg[i] ) += staticQPoints[ j ].h * staticQPoints[ j ].w;
       }  
      }            
		}
	}



}


__device__ void pseudoGsolMoving(int *offset, double *trans, double *pGsolMoving, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree, QPOINT_CUDA *movingQPoints, int *numMovingQPointsOctreeNodes, grid_cuda *staticPG, double *distanceCutoff, double *TRANSLATE, double *DIM, int *rangeCount)
{



	int i = threadIdx.x;


	if (i >= 0 && i < *numMovingQPointsOctreeNodes)
	{

		pGsolMoving[i] = 0;
		pGsolHMovingPos[i] = 0;
		pGsolHMovingNeg[i] = 0;

		if ( movingQPointsOctreeFlags[ *offset + i ])
		{
			for (int j = movingQPointsOctree[ i ].qPtsStartID;  j <= movingQPointsOctree[ i ].qPtsEndID; j++ )
			{
				Point_cuda p, q;
               
        p.x = movingQPoints[ j ].x;
        p.y = movingQPoints[ j ].y;
        p.z = movingQPoints[ j ].z;
               
        transformPoint_cuda( &p, trans, &q );
           
        if ( pointsWithinRange_cuda( &q, distanceCutoff, TRANSLATE, DIM, rangeCount, staticPG ) )
        {
        	( pGsolMoving[i] ) += movingQPoints[ j ].w;

          if ( movingQPoints[ j ].h > 0 ) 
						( pGsolHMovingPos[i] ) += movingQPoints[ j ].h * movingQPoints[ j ].w;
          else 
						( pGsolHMovingNeg[i] ) += movingQPoints[ j ].h * movingQPoints[ j ].w;
        }  
     }            
	
		}

	}

}





/*

__device__ void block_sum(double *input, double *results, int n)
{
//	printf("OLAAAAAA11111\n");
	extern __shared__ double sdata[];
	int i = threadIdx.x, tx = threadIdx.x;

	double x = 0;

	if (i < n)
		x = input[i];

	sdata[tx] = x;
	//printf("sdata[tx] %lf %d\n", sdata[tx], tx);
	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if (tx < offset)
			sdata[tx] += sdata[tx+offset];

		__syncthreads;
	}

//	__shared__ double *b;
	if (threadIdx.x == 0){
		//*results = sdata[0];
		printf("OLAAAAAA %lf\n", sdata[0]);//, *b);
	}
//	*results = sdata[0];
//	printf("OLAAAAAA %lf\n", *results);
	//results[blockIdx.x] = sdata[0];

}
*/


__global__ void kern1(/*static stuff*/ball_cuda *ballsStatic, int *totBallsStatic, int *offsetStatic, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes,  double *TRANSLATEstatic, double *DIMstatic, int *rangeCountStatic, double *distanceCutoff /*moving stuff*/, ball_cuda *ballsMoving, int *totBallsMoving, int *offsetMoving, double *trans, double *pGsolMoving, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree, QPOINT_CUDA *movingQPoints, int *numMovingQPointsOctreeNodes,  double *TRANSLATEmoving, double *DIMmoving, int *rangeCountMoving /*others*/)//, double *pGsol_scalar_d, double *pGsolHStaticPos_scalar_d, double *pGsolHStaticNeg_scalar_d, double *pGsolHMovingPos_scalar_d, double *pGsolHMovingNeg_scalar_d)
//__global__ void  kern(struct static_params *statP)
{
		//printf("ola\n");



			__shared__ grid_cuda movingGrid;

			if (threadIdx.x == 0)
				movingGrid = copyGrid(ballsStatic, totBallsStatic);

			__syncthreads();

			pseudoGsolStatic(&movingGrid, offsetStatic, transI, pGsolStatic, pGsolHStaticPos, pGsolHStaticNeg, staticQPointsOctreeFlags, staticQPointsOctree, staticQPoints, numStaticQPointsOctreeNodes, TRANSLATEstatic, DIMstatic, rangeCountStatic, distanceCutoff);




		}


__global__ void kern2(/*static stuff*/ball_cuda *ballsStatic, int *totBallsStatic, int *offsetStatic, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes,  double *TRANSLATEstatic, double *DIMstatic, int *rangeCountStatic, double *distanceCutoff /*moving stuff*/, ball_cuda *ballsMoving, int *totBallsMoving, int *offsetMoving, double *trans, double *pGsolMoving, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree, QPOINT_CUDA *movingQPoints, int *numMovingQPointsOctreeNodes,  double *TRANSLATEmoving, double *DIMmoving, int *rangeCountMoving /*others*/)//, double *pGsol_scalar_d, double *pGsolHStaticPos_scalar_d, double *pGsolHStaticNeg_scalar_d, double *pGsolHMovingPos_scalar_d, double *pGsolHMovingNeg_scalar_d)
//__global__ void  kern(struct static_params *statP)
{
		//printf("ola\n");



			__shared__ grid_cuda staticGrid;

			if (threadIdx.x == 0)
				staticGrid = copyGrid(ballsMoving, totBallsMoving);

			__syncthreads();

			pseudoGsolMoving(offsetMoving, trans, pGsolMoving, pGsolHMovingPos, pGsolHMovingNeg, movingQPointsOctreeFlags, movingQPointsOctree, movingQPoints, numMovingQPointsOctreeNodes, &staticGrid, distanceCutoff, TRANSLATEmoving, DIMmoving, rangeCountMoving);

}

/*




double sum(double *d_input, int n, int block_size, int num_blocks)
{
	double *d_sums = 0;
	hipMalloc((void**)&d_sums, sizeof(double));// * (num_Blocks+1));
	
	//int smem_sz = block_size* sizeof(double));

//	block_sum<<<num_blocks, block_size>>>(d_input);//, d_sums, n);
	block_sum<<<1,1>>>();//d_input);//, d_sums, n);
//	block_sum<<<1, block_size, smem_sz>>>(d_sums, dsums+num_blocks, num_blocks);

	double result = 0;
	hipMemcpy(&result, d_sums, sizeof(double),hipMemcpyDeviceToHost);
	
	return result;
}


*/
int pseudoGsol::call_to_kern(int offsetStatic, double *transI, double *pGsol, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE *staticQPointsOctree, QPOINT *staticQPoints, int numStaticQPoints, int numStaticQPointsOctreeNodes, PG *movingPG, double TRANSLATEstatic, double DIMstatic, int rangeCountStatic, PARAMS_IN params, int offsetMoving, double *trans, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE *movingQPointsOctree, QPOINT *movingQPoints, int numMovingQPoints, int numMovingQPointsOctreeNodes, PG *staticPG, double TRANSLATEmoving, double DIMmoving, int rangeCountMoving)
//int pseudoGsol::call_to_kern( PG *movingPG)
{

// convert movingGrid

		vector<ball_cuda> ballsVecMoving_h = convertGrid(movingPG);

		int totBallsMoving = ballsVecMoving_h.size();
		int *totBallsMoving_d;

		int totSizeMoving = totBallsMoving * (sizeof(int)*3+sizeof(Point));

		ball_cuda *arrayMoving_h = (ball_cuda*) malloc(totSizeMoving);
		std::copy(ballsVecMoving_h.begin(), ballsVecMoving_h.end(), arrayMoving_h);
		ball_cuda *arrayMoving_d;


// convert staticGrid

		vector<ball_cuda> ballsVecStatic_h = convertGrid(staticPG);

		int totBallsStatic = ballsVecStatic_h.size();
		int *totBallsStatic_d;

		int totSizeStatic = totBallsStatic * (sizeof(int)*3+sizeof(Point));

		ball_cuda *arrayStatic_h = (ball_cuda*) malloc(totSizeStatic);
		std::copy(ballsVecStatic_h.begin(), ballsVecStatic_h.end(), arrayStatic_h);
		ball_cuda *arrayStatic_d;



// passing structs


		struct QPOINT_CUDA *staticQPoints_h = (QPOINT_CUDA*) malloc(numStaticQPoints * sizeof(QPOINT_CUDA));
		struct QPOINT_CUDA *staticQPoints_d;

		for (int i = 0; i < numStaticQPoints; i++)
		{
			staticQPoints_h[i].x = staticQPoints[i].x;
			staticQPoints_h[i].y = staticQPoints[i].y;
			staticQPoints_h[i].z = staticQPoints[i].z;
			staticQPoints_h[i].w = staticQPoints[i].w;
			staticQPoints_h[i].h = staticQPoints[i].h;
		}			
		



		struct QPOINT_CUDA *movingQPoints_h = (QPOINT_CUDA*) malloc(numMovingQPoints * sizeof(QPOINT_CUDA));
		struct QPOINT_CUDA *movingQPoints_d;

		for (int i = 0; i < numMovingQPoints; i++)
		{
			movingQPoints_h[i].x = movingQPoints[i].x;
			movingQPoints_h[i].y = movingQPoints[i].y;
			movingQPoints_h[i].z = movingQPoints[i].z;
			movingQPoints_h[i].w = movingQPoints[i].w;
			movingQPoints_h[i].h = movingQPoints[i].h;
		}
		

// -------------------------

		struct QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree_h = (QPOINTS_OCTREE_NODE_CUDA*) malloc(numStaticQPointsOctreeNodes * sizeof(QPOINTS_OCTREE_NODE_CUDA));
		struct QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree_d;

		for (int i = 0; i < numStaticQPointsOctreeNodes; i++)
		{
			staticQPointsOctree_h[i].qPtsStartID = staticQPointsOctree[i].qPtsStartID;
			staticQPointsOctree_h[i].qPtsEndID = staticQPointsOctree[i].qPtsEndID;
		}



		struct QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree_h = (QPOINTS_OCTREE_NODE_CUDA*) malloc(numMovingQPointsOctreeNodes * sizeof(QPOINTS_OCTREE_NODE_CUDA));
		struct QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree_d;

		for (int i = 0; i < numMovingQPointsOctreeNodes; i++)
		{
			movingQPointsOctree_h[i].qPtsStartID = movingQPointsOctree[i].qPtsStartID;
			movingQPointsOctree_h[i].qPtsEndID = movingQPointsOctree[i].qPtsEndID;
		}
		

/*
hipError_t hipMemcpyAsync 	( 	void *  	dst,
		const void *  	src,
		size_t  	count,
		enum hipMemcpyKind  	kind,
		hipStream_t  	stream = 0	 
	) 
*/


		bool *staticQPointsOctreeFlags_d, *movingQPointsOctreeFlags_d;
		double *trans_d, *transI_d;

		double *pGsolStatic_h, *pGsolStatic_d, *pGsolMoving_h, *pGsolMoving_d;
		pGsolStatic_h = (double*) malloc(sizeof(double) * numStaticQPointsOctreeNodes);
		pGsolMoving_h = (double*) malloc(sizeof(double) * numMovingQPointsOctreeNodes);

		double *pGsolHStaticPos_h, *pGsolHStaticPos_d, *pGsolHStaticNeg_h, *pGsolHStaticNeg_d;
		pGsolHStaticPos_h = (double*) malloc(sizeof(double) * numStaticQPointsOctreeNodes);
		pGsolHStaticNeg_h = (double*) malloc(sizeof(double) * numStaticQPointsOctreeNodes);

		double *pGsolHMovingPos_h, *pGsolHMovingPos_d, *pGsolHMovingNeg_h, *pGsolHMovingNeg_d;
		pGsolHMovingPos_h = (double*) malloc(sizeof(double) * numMovingQPointsOctreeNodes);
		pGsolHMovingNeg_h = (double*) malloc(sizeof(double) * numMovingQPointsOctreeNodes);


		struct static_params statS_h, *statS_d;
		
		statS_h.offset = offsetStatic;
		statS_h.numStaticQPointsOctreeNodes = numStaticQPointsOctreeNodes;
		statS_h.distanceCutoff = params.distanceCutoff;
		statS_h.TRANSLATE = TRANSLATEstatic;
		statS_h.DIM = DIMstatic;
		statS_h.rangeCount = rangeCountStatic;


		struct moving_params statM_h, *statM_d;
		
		statM_h.offset = offsetMoving;
		statM_h.numMovingQPointsOctreeNodes = numMovingQPointsOctreeNodes;
		statM_h.distanceCutoff = params.distanceCutoff;
		statM_h.TRANSLATE = TRANSLATEmoving;
		statM_h.DIM = DIMmoving;
		statM_h.rangeCount = rangeCountMoving;

// pG values-----------------------

		//*pGsol = *pGsolHStaticPos = *pGsolHStaticNeg = *pGsolHMovingPos = *pGsolHMovingNeg = 0;

	//	double *pGsol_scalar_d, *pGsolHStaticPos_scalar_d, *pGsolHStaticNeg_scalar_d, *pGsolHMovingPos_scalar_d, *pGsolHMovingNeg_scalar_d;




// hipMalloc ---------------------------

		hipDeviceSetLimit (hipLimitMallocHeapSize, 512*1024*1024);

		hipMalloc( (void **)&arrayMoving_d, totSizeMoving );
		hipMalloc( (void **)&totBallsMoving_d, sizeof(int) );

		hipMalloc( (void **)&arrayStatic_d, totSizeStatic );
		hipMalloc( (void **)&totBallsStatic_d, sizeof(int) );


		hipMalloc( (void **)&staticQPoints_d, sizeof(QPOINT_CUDA)*numStaticQPoints );
		hipMalloc( (void **)&movingQPoints_d, sizeof(QPOINT_CUDA)*numMovingQPoints );

		hipMalloc( (void **)&staticQPointsOctree_d, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numStaticQPointsOctreeNodes );
		hipMalloc( (void **)&movingQPointsOctree_d, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numMovingQPointsOctreeNodes );

		hipMalloc( (void **)&staticQPointsOctreeFlags_d, sizeof(bool)*numStaticQPointsOctreeNodes );
		hipMalloc( (void **)&movingQPointsOctreeFlags_d, sizeof(bool)*numMovingQPointsOctreeNodes );

		hipMalloc( (void **)&trans_d, sizeof(double)*12 );
		hipMalloc( (void **)&transI_d, sizeof(double)*12 );

		hipMalloc( (void **)&statS_d, 3*(sizeof(int)+sizeof(double)) );//sizeof(static_params) );
		hipMalloc( (void **)&statM_d, 3*(sizeof(int)+sizeof(double)) );

		
		hipMalloc( (void **)&pGsolStatic_d, sizeof(double) * numStaticQPointsOctreeNodes);
		hipMalloc( (void **)&pGsolMoving_d, sizeof(double) * numMovingQPointsOctreeNodes);

		hipMalloc( (void **)&pGsolHStaticPos_d, sizeof(double) * numStaticQPointsOctreeNodes);
		hipMalloc( (void **)&pGsolHStaticNeg_d, sizeof(double) * numStaticQPointsOctreeNodes);

		hipMalloc( (void **)&pGsolHMovingPos_d, sizeof(double) * numMovingQPointsOctreeNodes);
		hipMalloc( (void **)&pGsolHMovingNeg_d, sizeof(double) * numMovingQPointsOctreeNodes);


	//	hipMalloc( (void **)&pGsol_scalar_d, sizeof(double));

	//	hipMalloc( (void **)&pGsolHStaticPos_scalar_d, sizeof(double));
	//	hipMalloc( (void **)&pGsolHStaticNeg_scalar_d, sizeof(double));

	//	hipMalloc( (void **)&pGsolHMovingPos_scalar_d, sizeof(double));
	//	hipMalloc( (void **)&pGsolHMovingNeg_scalar_d, sizeof(double));


		//int *hist_h = (int*) malloc(sizeof(int)*numStaticQPointsOctreeNodes);
//		int *hist_d;
		//hipMalloc( (void **)&hist_d, sizeof(int) * numStaticQPointsOctreeNodes);

// hipMemcpy-------------------

		hipMemcpy( arrayMoving_d, arrayMoving_h, totSizeMoving, hipMemcpyHostToDevice );
		hipMemcpy( totBallsMoving_d, &totBallsMoving, sizeof(int), hipMemcpyHostToDevice );

		hipMemcpy( arrayStatic_d, arrayStatic_h, totSizeStatic, hipMemcpyHostToDevice );
		hipMemcpy( totBallsStatic_d, &totBallsStatic, sizeof(int), hipMemcpyHostToDevice );

		hipMemcpy( staticQPoints_d, staticQPoints_h, sizeof(QPOINT_CUDA)*numStaticQPoints, hipMemcpyHostToDevice );
		hipMemcpy( movingQPoints_d, movingQPoints_h, sizeof(QPOINT_CUDA)*numMovingQPoints, hipMemcpyHostToDevice );

		hipMemcpy( staticQPointsOctree_d, staticQPointsOctree_h, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numStaticQPointsOctreeNodes, hipMemcpyHostToDevice );
		hipMemcpy( movingQPointsOctree_d, movingQPointsOctree_h, sizeof(QPOINTS_OCTREE_NODE_CUDA)*numMovingQPointsOctreeNodes, hipMemcpyHostToDevice );

	

		hipMemcpy( staticQPointsOctreeFlags_d, staticQPointsOctreeFlags, sizeof(bool)*numStaticQPointsOctreeNodes, hipMemcpyHostToDevice );
		hipMemcpy( movingQPointsOctreeFlags_d, movingQPointsOctreeFlags, sizeof(bool)*numMovingQPointsOctreeNodes, hipMemcpyHostToDevice );


		hipMemcpy( trans_d, trans, sizeof(double)*12, hipMemcpyHostToDevice );
		hipMemcpy( transI_d, transI, sizeof(double)*12, hipMemcpyHostToDevice );


		hipMemcpy( statS_d, &statS_h, 3*(sizeof(int)+sizeof(double)), hipMemcpyHostToDevice );
		hipMemcpy( statM_d, &statM_h, 3*(sizeof(int)+sizeof(double)), hipMemcpyHostToDevice );


// call Kernel ---------------------

		int numBlocks = 1;
		int threadsPerBlock;

		threadsPerBlock = numStaticQPointsOctreeNodes;

//		int threadsPerBlock = 1;

		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error1: %s\n", hipGetErrorString(err));

		printf("calling... %d\n", threadsPerBlock);

		kern1<<< numBlocks,threadsPerBlock >>>(arrayMoving_d, totBallsMoving_d, &(statS_d->offset), transI_d, pGsolStatic_d, pGsolHStaticPos_d, pGsolHStaticNeg_d, staticQPointsOctreeFlags_d, staticQPointsOctree_d, staticQPoints_d, &(statS_d->numStaticQPointsOctreeNodes), &(statS_d->TRANSLATE), &(statS_d->DIM), &(statS_d->rangeCount), &(statS_d->distanceCutoff), arrayStatic_d, totBallsStatic_d, &(statM_d->offset), trans_d, pGsolMoving_d, pGsolHMovingPos_d, pGsolHMovingNeg_d, movingQPointsOctreeFlags_d, movingQPointsOctree_d, movingQPoints_d, &(statM_d->numMovingQPointsOctreeNodes), &(statM_d->TRANSLATE), &(statM_d->DIM), &(statM_d->rangeCount));

	threadsPerBlock = numMovingQPointsOctreeNodes;

	kern2<<< numBlocks,threadsPerBlock >>>(arrayMoving_d, totBallsMoving_d, &(statS_d->offset), transI_d, pGsolStatic_d, pGsolHStaticPos_d, pGsolHStaticNeg_d, staticQPointsOctreeFlags_d, staticQPointsOctree_d, staticQPoints_d, &(statS_d->numStaticQPointsOctreeNodes), &(statS_d->TRANSLATE), &(statS_d->DIM), &(statS_d->rangeCount), &(statS_d->distanceCutoff), arrayStatic_d, totBallsStatic_d, &(statM_d->offset), trans_d, pGsolMoving_d, pGsolHMovingPos_d, pGsolHMovingNeg_d, movingQPointsOctreeFlags_d, movingQPointsOctree_d, movingQPoints_d, &(statM_d->numMovingQPointsOctreeNodes), &(statM_d->TRANSLATE), &(statM_d->DIM), &(statM_d->rangeCount));

//(/*static stuff*/ball_cuda *ballsStatic, int *totBallsStatic, int *offsetStatic, double *transI, double *pGsolStatic, double *pGsolHStaticPos, double *pGsolHStaticNeg, bool *staticQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *staticQPointsOctree, QPOINT_CUDA *staticQPoints, int *numStaticQPointsOctreeNodes,  double *TRANSLATEstatic, double *DIMstatic, int *rangeCountStatic, double *distanceCutoff /*moving stuff*/, ball_cuda *ballsMoving, int *totBallsMoving, int *offsetMoving, double *trans, double *pGsolMoving, double *pGsolHMovingPos, double *pGsolHMovingNeg, bool *movingQPointsOctreeFlags, QPOINTS_OCTREE_NODE_CUDA *movingQPointsOctree, QPOINT_CUDA *movingQPoints, int *numMovingQPointsOctreeNodes,  double *TRANSLATEmoving, double *DIMmoving, int *rangeCountMoving /*others*/, double *pGsol_scalar_d, double *pGsolHStaticPos_scalar_d, double *pGsolHStaticNeg_scalar_d, double *pGsolHMovingPos_scalar_d, double *pGsolHMovingNeg_scalar_d)
//__global__ void  kern(struct static_params *statP)


//, pGsol_scalar_d, pGsolHStaticPos_scalar_d, pGsolHStaticNeg_scalar_d, pGsolHMovingPos_scalar_d, pGsolHMovingNeg_scalar_d);



		printf("end...\n");

		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error2: %s\n", hipGetErrorString(err));


		for (int i = 0; i < numStaticQPointsOctreeNodes; i++){
			pGsolStatic_h[i] = 0;
			pGsolHStaticPos_h[i] = 0;
			pGsolHStaticNeg_h[i] = 0;
		}


		for (int i = 0; i < numMovingQPointsOctreeNodes; i++){
			pGsolMoving_h[i] = 0;
			pGsolHMovingPos_h[i] = 0;
			pGsolHMovingNeg_h[i] = 0;
		}


		hipMemcpy( pGsolStatic_h, pGsolStatic_d, sizeof(double)* numStaticQPointsOctreeNodes, hipMemcpyDeviceToHost );
		hipMemcpy( pGsolHStaticPos_h, pGsolHStaticPos_d, sizeof(double)* numStaticQPointsOctreeNodes, hipMemcpyDeviceToHost );
		hipMemcpy( pGsolHStaticNeg_h, pGsolHStaticNeg_d, sizeof(double)* numStaticQPointsOctreeNodes, hipMemcpyDeviceToHost );	


		hipMemcpy( pGsolMoving_h, pGsolMoving_d, sizeof(double)* numMovingQPointsOctreeNodes, hipMemcpyDeviceToHost );
		hipMemcpy( pGsolHMovingPos_h, pGsolHMovingPos_d, sizeof(double)* numMovingQPointsOctreeNodes, hipMemcpyDeviceToHost );
		hipMemcpy( pGsolHMovingNeg_h, pGsolHMovingNeg_d, sizeof(double)* numMovingQPointsOctreeNodes, hipMemcpyDeviceToHost );	


		//hipMemcpy( hist_h, hist_d, sizeof(int)*numStaticQPointsOctreeNodes, hipMemcpyDeviceToHost );


		err = hipGetLastError();
		if (err != hipSuccess) 
		  printf("Error3: %s\n", hipGetErrorString(err));


// do reductions 
		double sPgSt = 0;
		double sPosSt = 0;
		double sNegSt = 0;

		for (int i = 0; i < numStaticQPointsOctreeNodes; i++){
			sPgSt += pGsolStatic_h[i];
			sPosSt += pGsolHStaticPos_h[i];
			sNegSt += pGsolHStaticNeg_h[i];
		}

		double sPgMov = 0;
		double sPosMov = 0;
		double sNegMov = 0;

		for (int i = 0; i < numMovingQPointsOctreeNodes; i++){
			sPgMov += pGsolMoving_h[i];
			sPosMov += pGsolHMovingPos_h[i];
			sNegMov += pGsolHMovingNeg_h[i];
		}



printf("CUDA *pGsol, *pGsolHStaticPos, *pGsolHStaticNeg, *pGsolHMovingPos, *pGsolHMovingNeg %lf %lf %lf %lf %lf\n", sPgSt+sPgMov, sPosSt, sNegSt, sPosMov, sNegMov);




		hipFree(arrayMoving_d);
		hipFree(totBallsMoving_d);
		hipFree(arrayStatic_d);
		hipFree(totBallsStatic_d);

		hipFree(staticQPoints_d);
		hipFree(movingQPoints_d);

		hipFree(staticQPointsOctree_d);
		hipFree(movingQPointsOctree_d);

		hipFree(staticQPointsOctreeFlags_d);
		hipFree(movingQPointsOctreeFlags_d);

		hipFree(trans_d);
		hipFree(transI_d);

		hipFree(statS_d);
		hipFree(statM_d);

		hipFree(pGsolStatic_d);
		hipFree(pGsolMoving_d);

		hipFree(pGsolHStaticPos_d);
		hipFree(pGsolHStaticNeg_d);

		hipFree(pGsolHMovingPos_d);
		hipFree(pGsolHMovingNeg_d);


		free(arrayMoving_h);
		free(arrayStatic_h);

		free(pGsolStatic_h);
		free(pGsolMoving_h);

		free(pGsolHStaticPos_h);
		free(pGsolHStaticNeg_h);
		free(pGsolHMovingPos_h);
		free(pGsolHMovingNeg_h);

		free(staticQPointsOctree_h);
		free(staticQPoints_h);

		free(movingQPointsOctree_h);
		free(movingQPoints_h);

		hipDeviceReset();

	return 0;
}




vector<ball_cuda> convertGrid(PG *grid_)
{

		int nPlanes;
		int nLines;
		int nCells;
		int nBalls;

		int grid_init, grid_end;	
		int plane_init, plane_end;
		int line_init, line_end;
		
		vector<tuple<plane*> > grid1;

		tuple<plane*> plane1;
		vector<tuple<line*> > plane2;

		tuple<line*> line1;
		vector<tuple<gridcell*> > line2;
		
		tuple<gridcell*> cell1;
		
		vector<ball_cuda> ballsVec_h;
		ball_cuda tempBall_h;

		grid g1 = grid_->getGrid();

		nPlanes = g1.RR.getn();


		grid_init = g1.RR.getOverallMin();
		grid_end = g1.RR.getOverallMax();	
		grid1 = g1.RR.report(grid_init, grid_end);

		
		// goes through all planes
		for (int i = 0; i < nPlanes; i++)
		{
			plane1 = grid1.at(i);
	//		printf("plane ID %d\n", plane1.id);

			nLines = (plane1.ptr)->RR.getn();
//			printf(" nLines %d\n", nLines);
			
			plane_init =(plane1.ptr)->RR.getOverallMin(); 
			plane_end = (plane1.ptr)->RR.getOverallMax();

			plane2 = (plane1.ptr)->RR.report(plane_init, plane_end);

			// goes through all lines
			for (int j = 0; j < nLines; j++)
			{
				line1 = plane2.at(j);
	//			printf("	line ID %d\n", line1.id);

				nCells = (line1.ptr)->RR.getn();
		//		printf("nCells  %d\n", nCells);

				line_init = (line1.ptr)->RR.getOverallMin(); 
				line_end = (line1.ptr)->RR.getOverallMax(); 
				line2 = (line1.ptr)->RR.report(line_init, line_end);

				// goes through all cells
				for (int k = 0; k < nCells; k++)
				{
					cell1 = line2.at(k);
	//				printf("		cell ID %d\n", cell1.id);
					

					nBalls = cell1.ptr->balls.size();
	//				printf("			nBalls %d\n", nBalls);

					// create vector with total number of balls and plane, line, and cell ID
					//goes through all balls
					for (int kk = 0 ; kk < nBalls; kk++)
					{
					
						tempBall_h.cell_id = cell1.id;
						tempBall_h.parentLine = line1.id;
						tempBall_h.parentPlane = plane1.id;
						tempBall_h.ball = Point (cell1.ptr->balls.at(kk)->x, cell1.ptr->balls.at(kk)->y, cell1.ptr->balls.at(kk)->z);

						ballsVec_h.push_back(tempBall_h);

					} //end balls	
				} // end cells
			} // end lines
		} // end planes
		
		return ballsVec_h;

}
